#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "swap.h"
#include "state.h"
#include "utils.h"
#include "helper_cuda.hpp"
#include <pthread.h>
#include <algorithm>
void CSQS(const State &qureg,  const int csqsSize);
void CSQS(const State &qureg,  const int csqsSize, char* targ);
void mapping_after_swap_conti(const State& qureg, Fp* graph,int SwapOut,int SwapIn,int numSwap){
    assert(SwapIn > SwapOut);
    assert(SwapIn + numSwap -1 <= qureg.numQubits);
    int num = qureg.numQubits;
    while(numSwap>0){
        for(int i = 0 ; i < num ;i++){
            if(i == SwapIn || i == SwapOut)continue;
            else if(i < SwapOut){
                std::swap(graph[num * i + SwapIn], graph[num*i+SwapOut]);
            }
            else if(i > SwapOut && i < SwapIn){
                std::swap(graph[num*SwapOut+i], graph[num*i+SwapIn]);
            }
            else{
                std::swap(graph[num * SwapOut+i], graph[num * SwapIn+i]);
            }
        }
        numSwap--;
        SwapIn++;
        SwapOut++;
    }

    
}


__device__ __forceinline__
ull bit_string(const ull task, const int targ0, const int targ1)
{
	ull res = task;
	ull mask;
	mask = (1ULL << targ0) - 1;
	res = ((res >> targ0) << (targ0+1)) | (res & mask);
	mask = (1ULL << targ1) - 1;
	res = ((res >> targ1) << (targ1+1)) | (res & mask);
    return res;
}

__device__ __forceinline__
long long bit_string_multi(const long long task, const int targ, const int amount)
{
	long long res = task;
	long long mask;
	mask = (1ULL << targ) - 1;
	res = ((res >> targ) << (targ+amount)) | (res & mask);
    return res;
}


__global__
void _swap_gate_conti(Complex* sv, const  int swapOut, const int swapIn, const int numSwap, const int* d_table){
    ull tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int numSwapBits = (numSwap<<1)-1; 

    size_t gidx = tidx >> numSwapBits;

    gidx = bit_string_multi(gidx, swapOut, numSwap);//Swapout < Swapin
    gidx = bit_string_multi(gidx, swapIn, numSwap);


    int swapIdx = tidx & ((1<<numSwapBits)-1);

    if (swapIdx >= ((1<<numSwap) * ((1<<numSwap)-1) / 2))
    return;

    size_t s1 = d_table[2*swapIdx];
    size_t s2 = d_table[2*swapIdx+1];

    size_t up = (s1 << swapIn) + (s2 << swapOut) + gidx;
    size_t lo = (s1 << swapOut) + (s2 << swapIn) + gidx;

    Complex tmp = sv[up];
    sv[up] = sv[lo];
    sv[lo] = tmp;
}

void swap_gate_conti(const State& qureg, const int SwapOut, const int SwapIn, int numSwap){
        assert(SwapIn > SwapOut);
        if(qureg.numDevice > 1 && SwapIn + numSwap -1 >= qureg.numQubitsPerDevice){
            int numQubitOutofDevice = SwapIn + numSwap - qureg.numQubitsPerDevice; //number of qubit out of device
            if(SwapIn < qureg.numQubitsPerDevice){
                swap_gate_conti(qureg,  
                                qureg.numQubitsPerDevice - SwapIn, 
                                qureg.numQubitsPerDevice - numQubitOutofDevice, 
                                numQubitOutofDevice);
                char* targ = (char*)malloc((SwapIn + numSwap - qureg.numQubitsPerDevice) * sizeof(char));
                for(int i = 0;i< SwapIn + numSwap - qureg.numQubitsPerDevice ;i++){
                    targ[i] = i;
                }
                
                hipEvent_t start_event, end_event;
                hipEventCreate(&start_event);
                hipEventCreate(&end_event);
                hipEventRecord(start_event, qureg.gpus->compute_stream);

                CSQS(qureg, SwapIn + numSwap - qureg.numQubitsPerDevice, targ);

                hipEventRecord(end_event, qureg.gpus->compute_stream);
                hipEventSynchronize(end_event);
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start_event, end_event);
                csqs_time_ms += milliseconds;
                hipEventDestroy(start_event);
                hipEventDestroy(end_event);

                swap_gate_conti(qureg, 
                                qureg.numQubitsPerDevice - SwapIn, 
                                qureg.numQubitsPerDevice - numQubitOutofDevice, 
                                numQubitOutofDevice);
            }
            else{
                swap_gate_conti(qureg, 0, qureg.numQubitsPerDevice - numSwap, numSwap);
                char* targ = (char*)malloc(numSwap * sizeof(char));
                for(int i = 0;i< numSwap ;i++){
                    targ[i] = SwapIn + i - qureg.numQubitsPerDevice;
                }
                
                hipEvent_t start_event, end_event;
                hipEventCreate(&start_event);
                hipEventCreate(&end_event);
                hipEventRecord(start_event, qureg.gpus->compute_stream);

                CSQS(qureg, numSwap, targ);

                hipEventRecord(end_event, qureg.gpus->compute_stream);
                hipEventSynchronize(end_event);
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start_event, end_event);
                csqs_time_ms += milliseconds;
                hipEventDestroy(start_event);
                hipEventDestroy(end_event);

                swap_gate_conti(qureg, 0, qureg.numQubitsPerDevice - numSwap, numSwap);
                return;
            }
            numSwap -= numQubitOutofDevice; 
            if(!numSwap)return;;
        }
        assert(SwapIn + numSwap -1 <= qureg.numQubitsPerDevice);
        ull grid = 1;
        ull block = qureg.numAmpsPerDevice >> 1;
        if (block > 512) {
            grid = block / 512 ;
            block = 512;
        }
		ull numSwapPairs = (1ull<<numSwap) * ((1ull<<numSwap)-1) / 2;
        ull swap_table_size = numSwapPairs * 2 * sizeof(int);
		int *table;

		table = (int*) malloc(swap_table_size);

		int counter = 0;
		for (int i=0; i<(1<<numSwap); i++)
			for (int j=i+1; j<(1<<numSwap); j++, counter++) {
				table[2*counter] = i;
				table[2*counter+1] = j;
			}

		assert(counter == numSwapPairs);

#if USE_MPI
        checkCudaErrors(hipMemcpyAsync(qureg.gpus->d_table, table, swap_table_size, hipMemcpyHostToDevice, qureg.gpus->compute_stream));
        // checkCudaErrors(hipMemcpy(qureg.gpus->d_table, table, swap_table_size, hipMemcpyHostToDevice));
        _swap_gate_conti<<<grid, block, 0 ,qureg.gpus->compute_stream>>>(qureg.gpus->dState, SwapOut, SwapIn, numSwap, qureg.gpus->d_table);

#else
        for (int dev = 0; dev < qureg.numDevice; dev++) {
            checkCudaErrors(hipSetDevice(dev));
            checkCudaErrors(hipMemcpyAsync(qureg.gpus[dev].d_table, table, swap_table_size, hipMemcpyHostToDevice, qureg.gpus[dev].compute_stream));
            _swap_gate_conti<<<grid, block, 0 ,qureg.gpus[dev].compute_stream>>>(qureg.gpus[dev].dState, SwapOut, SwapIn, numSwap, qureg.gpus[dev].d_table);
            // checkCudaErrors(hipFreeAsync(qureg.gpus[dev].d_table,qureg.gpus[dev].compute_stream));
        }
#endif

}

__host__ __device__ __forceinline__
uint64_t insert_bit_0(uint64_t task, const char targ)
{
	uint64_t mask = (1ULL << targ) - 1;
	return ((task >> targ) << (targ+1)) | (task & mask);
}


template<int N>
__host__ __device__ __forceinline__
uint64_t insert_bits_0(uint64_t task, const char targs[])
{
	#pragma unroll
	for (int i = 0; i < N; i++)
		task = insert_bit_0(task, targs[i]);
	return task;
}


void CSQS(const State &qureg,  const int csqsSize, char* targ)
{
    const int &N = qureg.numQubits;
    const int &D = qureg.numQubits - qureg.numQubitsPerDevice;
    const int &B = qureg.numQubitsPerBuffer;

    const int numGroups = 1 << (D - csqsSize);
    const int numMemberInGroup = 1 << csqsSize;

    std::vector<std::vector<int>> devlist(numGroups, std::vector<int>(numMemberInGroup));
    // Deal with all to all problem.
    for (int mbr = 0; mbr < numMemberInGroup; mbr++) {
        int mem_bits = 0;
        for (int i = 0; i < csqsSize; i++)
            mem_bits |= ((mbr >> i) & 1) << targ[i];
        devlist[0][mbr] = mem_bits;
    }
   
    for (int grp = 1; grp < (numGroups); grp++)
    {
        int grp_bits = grp;
        for (int i = 0; i < (D - csqsSize); i++)
            grp_bits = insert_bits_0<1>(grp_bits, &targ[i]);
        for (int mbr = 0; mbr < numMemberInGroup; mbr++)
            devlist[grp][mbr] = grp_bits | devlist[0][mbr];
    }

 
    // Using buffer to tackle ncclSend and ncclRecv
    for (ull off = 0; off < (1ull << (N - D - csqsSize)); off += (1 << (B - csqsSize)))
    {
        for (int grp = 0; grp < (1 << (D - csqsSize)); grp++)
        {   
            hipEvent_t transmission_start_event, transmission_end_event;
            hipEventCreate(&transmission_start_event);
            hipEventCreate(&transmission_end_event);
            hipEventRecord(transmission_start_event, qureg.gpus->compute_stream);

            checkCudaErrors(ncclGroupStart());
            for (int a = 0; a < (1 << csqsSize); a++)
            {
                int devA = devlist[grp][a];
#if USE_MPI
                if(qureg.gpus->world_rank == devA){
                    for (int b = 0; b < (1 << csqsSize); b++)
                    {
                        if (a == b)
                            continue;
                        int devB = devlist[grp][b];
                        int off_sv = b * (1ull << (N - D - csqsSize)) + off;
                        int off_bf = b * (1ull << (B - csqsSize));
                        
                        checkCudaErrors(ncclSend(qureg.gpus->dState + off_sv, (1 << (B - csqsSize)) * sizeof(Complex), ncclChar, devB, qureg.gpus->comm, qureg.gpus->compute_stream)); 
                        checkCudaErrors(ncclRecv(qureg.gpus->dBuf + off_bf, (1 << (B - csqsSize)) * sizeof(Complex), ncclChar, devB, qureg.gpus->comm, qureg.gpus->compute_stream));

                    }
                }

#else
                // checkCudaErrors(ncclGroupStart());
                for (int b = 0; b < (1 << csqsSize); b++)
                {
                    if (a == b)
                        continue;
                    int devB = devlist[grp][b];
                    int off_sv = b * (1ull << (N - D - csqsSize)) + off;
                    int off_bf = b * (1ull << (B - csqsSize));
                    
                    checkCudaErrors(ncclSend(qureg.gpus[devA].dState + off_sv, (1 << (B - csqsSize)) * sizeof(Complex), ncclChar, devB, qureg.gpus[devA].comm, qureg.gpus[devA].compute_stream)); 
                    checkCudaErrors(ncclRecv(qureg.gpus[devA].dBuf + off_bf, (1 << (B - csqsSize)) * sizeof(Complex), ncclChar, devB, qureg.gpus[devA].comm, qureg.gpus[devA].compute_stream));
                }

#endif
                // checkCudaErrors(ncclGroupEnd());
            }
            checkCudaErrors(ncclGroupEnd());

            hipEventRecord(transmission_end_event, qureg.gpus->compute_stream);
            hipEventSynchronize(transmission_end_event);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, transmission_start_event, transmission_end_event);
            transmission_time_ms += milliseconds;
            hipEventDestroy(transmission_start_event);
            hipEventDestroy(transmission_end_event);
}

        for (int grp = 0; grp < (1 << (D - csqsSize)); grp++)
        {
            // checkCudaErrors(ncclGroupStart());
            for (int a = 0; a < (1 << csqsSize); a++)
            {
                int devA = devlist[grp][a];
#if USE_MPI
                if(qureg.gpus->world_rank == devA){
                    for (int b = 0; b < (1 << csqsSize); b++)
                    {
                        if (a == b)
                            continue;
                        int off_sv = b * (1ull << (N - D - csqsSize)) + off;
                        int off_bf = b * (1ull << (B - csqsSize));
                        checkCudaErrors(hipMemcpyAsync(qureg.gpus->dState + off_sv, qureg.gpus->dBuf + off_bf, (1 << (B - csqsSize)) * sizeof(Complex), hipMemcpyDeviceToDevice, qureg.gpus->compute_stream));
                        // checkCudaErrors(hipMemcpy(qureg.gpus->dState + off_sv, qureg.gpus->dBuf + off_bf, (1 << (B - csqsSize)) * sizeof(Complex), hipMemcpyDeviceToDevice));
                    }
                }

#else
                // checkCudaErrors(ncclGroupStart());
                for (int b = 0; b < (1 << csqsSize); b++)
                {
                    if (a == b)
                        continue;
                    int off_sv = b * (1ull << (N - D - csqsSize)) + off;
                    int off_bf = b * (1ull << (B - csqsSize));
                    checkCudaErrors(hipMemcpyAsync(qureg.gpus[devA].dState + off_sv, qureg.gpus[devA].dBuf + off_bf, (1 << (B - csqsSize)) * sizeof(Complex), hipMemcpyDeviceToDevice, qureg.gpus[devA].compute_stream));
                }
#endif
                // checkCudaErrors(ncclGroupEnd());
            }
            // checkCudaErrors(ncclGroupEnd());
        }
    }

}